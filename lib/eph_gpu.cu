
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__
void dummy_test(int myID, int nrPS)
{
  int thread_index = threadIdx.x;
  int block_dimension = blockDim.x;
  int grid_dimension = gridDim.x;

  printf("GPU %d %d %d %d %d\n", myID, nrPS, thread_index, block_dimension, grid_dimension);
}

__global__
void calculate_environment_gpu_cu()
{
  
}


void run_dummy_test_cu(int myID, int nrPS) 
{
  dummy_test<<<1, 1>>>(myID, nrPS);
}

